/**
    Copyright (c) 2015 <wataro>
    
    This software is released under the MIT License.

    http://opensource.org/licenses/mit-license.php
*/
#include <hip/hip_runtime.h>

void * allocate_cuda_memory(size_t size)
{
    void * p;
    hipMalloc(&p, size);
    return p;
}

void delete_cuda_memory(void * p)
{
    hipFree(p);
}

